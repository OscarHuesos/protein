#include "hip/hip_runtime.h"
#include <iostream>
#include "clases.hpp"
#include <math.h>
#include <stdio.h>
#include <string.h>
#include <stdbool.h>

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

using namespace std;

void Proteina::apartar_lista_atomos(int n){
hipMallocManaged(&atomos_en_proteina,n*sizeof(Atomo));
}

void Proteina::destroy_list_obj_2(){
hipFree(atomos_en_proteina);
}

void Proteina::clear_interface_list(){
hipDeviceReset();
}

__global__ void Inter(Atomo* atomos,int No_atomos,float tol){

  int thx= threadIdx.x;
  int thy= threadIdx.y;
  int bx= blockIdx.x;
  int by= blockIdx.y;
  //coordernadas del hilo
  int nBlocks= gridDim.x * gridDim.y;
  //per bloque
  int nThreads= blockDim.x*blockDim.y;
  int idThread= (thy*blockDim.x + thx);
  int idBlock= (by*gridDim.x + bx);
  int uniqueThread= nThreads*idBlock + idThread;
  float distancia;
  float r=0;

if(uniqueThread<No_atomos){

for(int i=0;i< No_atomos;i++){
if(atomos[uniqueThread].chain_id!=atomos[i].chain_id){
distancia=atomos[uniqueThread].radii+atomos[i].radii+tol;
r=sqrt(pow(atomos[uniqueThread].x-atomos[i].x,2)+pow(atomos[uniqueThread].y-atomos[i].y,2)+pow(atomos[uniqueThread].z-atomos[i].z,2));

if(r<=distancia){
atomos[uniqueThread].interfaz=true;
break;
}
}else{
//
}
}
//}
}else{
//se_paso=true;
}

}

void Proteina::interface(){
int a;
int cadena_residuo;
int count=0;
float tolerancia_interfaz=0.5;
apartar_lista_atomos(No_atomos_en_prot);
//int size_char=0;
for(int i=0; i< No_residuos;i++){
a=residuos_en_proteina[i].No_atomos;
cadena_residuo=residuos_en_proteina[i].id_cadena;
for(int j=0; j< a;j++){
residuos_en_proteina[i].atomos_en_residuo[j].cadena_identificador(cadena_residuo);
atomos_en_proteina[count]=residuos_en_proteina[i].atomos_en_residuo[j];
count++;
}
}

dim3 dimGrid(30,30);
dim3 dimBlock(32,16);
printf("cont %d \n",count );
printf("atomos %d \n",No_atomos_en_prot );

Inter<<<dimGrid,dimBlock>>>(atomos_en_proteina,No_atomos_en_prot,tolerancia_interfaz);
hipDeviceSynchronize();
cudaCheckErrors("kernel fail");
int buffer;
int s;
for(int k=0; k < No_atomos_en_prot; ++k){
if(atomos_en_proteina[k].interfaz==true) {
buffer=atomos_en_proteina[k].Id_residuo;

for(int j=0;j<No_residuos;j++){
s=residuos_en_proteina[j].Id_Res;
if(buffer==s){
residuos_en_proteina[j].interfaz=true;
}
}
//printf("atomo de interfaz del residuo %d \n",atomos_en_proteina[k].Id_residuo);
}
}

for(int n=0;n<No_residuos;n++){
if(residuos_en_proteina[n].interfaz==true){
printf("residuo %s de interfaz ID %d en cadena %s \n",residuos_en_proteina[n].Nombre_res,residuos_en_proteina[n].Id_Res,residuos_en_proteina[n].Nombre_cad);
}
}

destroy_list_obj_2();
}
